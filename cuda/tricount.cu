#include "hip/hip_runtime.h"
#include <cstdlib>

#include "log.h"
#include "util.h"

void InitCuda() {
  hipSetDevice(0);
  hipFree(0);
}

__global__ void CUDAGetEdgeSup(EdgeT halfEdgesNum, const NodeT *halfEdgesFirst, const NodeT *halfEdgesSecond,
                               const EdgeT *halfNodeIndex, NodeT *edgesSup) {
  auto from = blockDim.x * blockIdx.x + threadIdx.x;
  auto step = gridDim.x * blockDim.x;
  for (EdgeT i = from; i < halfEdgesNum; i += step) {
    NodeT u = halfEdgesFirst[i];
    NodeT v = halfEdgesSecond[i];
    EdgeT uStart = halfNodeIndex[u];
    EdgeT uEnd = halfNodeIndex[u + 1];
    EdgeT vStart = halfNodeIndex[v];
    EdgeT vEnd = halfNodeIndex[v + 1];
    while (uStart < uEnd && vStart < vEnd) {
      if (halfEdgesSecond[uStart] < halfEdgesSecond[vStart]) {
        ++uStart;
      } else if (halfEdgesSecond[uStart] > halfEdgesSecond[vStart]) {
        ++vStart;
      } else {
        atomicAdd(edgesSup + i, 1);
        atomicAdd(edgesSup + uStart, 1);
        atomicAdd(edgesSup + vStart, 1);
        ++uStart;
        ++vStart;
      }
    }
  }
}

// 三角形计数获取支持边数量
void GetEdgeSup(EdgeT halfEdgesNum, const NodeT *halfEdgesFirst, const NodeT *halfEdgesSecond,
                const EdgeT *halfNodeIndex, NodeT nodesNum, NodeT *&edgesSup) {
  NodeT *cudaHalfEdgesFirst;
  NodeT *cudaHalfEdgesSecond;
  EdgeT *cudaHalfNodeIndex;
  NodeT *cudaEdgesSup;
  CUDA_TRY(hipMalloc((void **)&cudaHalfEdgesFirst, halfEdgesNum * sizeof(NodeT)));
  CUDA_TRY(hipMemcpy(cudaHalfEdgesFirst, halfEdgesFirst, halfEdgesNum * sizeof(NodeT), hipMemcpyHostToDevice));
  CUDA_TRY(hipMalloc((void **)&cudaHalfEdgesSecond, halfEdgesNum * sizeof(NodeT)));
  CUDA_TRY(hipMemcpy(cudaHalfEdgesSecond, halfEdgesSecond, halfEdgesNum * sizeof(NodeT), hipMemcpyHostToDevice));
  CUDA_TRY(hipMalloc((void **)&cudaHalfNodeIndex, (nodesNum + 1) * sizeof(EdgeT)));
  CUDA_TRY(hipMemcpy(cudaHalfNodeIndex, halfNodeIndex, (nodesNum + 1) * sizeof(EdgeT), hipMemcpyHostToDevice));
  CUDA_TRY(hipMalloc((void **)&cudaEdgesSup, halfEdgesNum * sizeof(NodeT)));
  log_info("1");
  CUDAGetEdgeSup<<<(halfEdgesNum + 127) / 128, 128>>>(halfEdgesNum, cudaHalfEdgesFirst, cudaHalfEdgesSecond,
                                                      cudaHalfNodeIndex, cudaEdgesSup);
  CUDA_TRY(hipDeviceSynchronize());
  edgesSup = (NodeT *)calloc(halfEdgesNum, sizeof(NodeT));
  CUDA_TRY(hipMemcpy(edgesSup, cudaEdgesSup, halfEdgesNum * sizeof(NodeT), hipMemcpyDeviceToHost));
  CUDA_TRY(hipDeviceSynchronize());

  CUDA_TRY(hipFree(cudaHalfEdgesFirst));
  CUDA_TRY(hipFree(cudaHalfEdgesSecond));
  CUDA_TRY(hipFree(cudaHalfNodeIndex));
  CUDA_TRY(hipFree(cudaEdgesSup));
  CUDA_TRY(hipDeviceSynchronize());
}
