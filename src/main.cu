#include "hip/hip_runtime.h"
#include "log.h"

__global__ void add(int a, int b, int *c) { *c = a + b; }

void init_gpu() {
  hipDeviceProp_t deviceProp{};
  hipGetDeviceProperties(&deviceProp, 0);
  uint64_t gpu_mem = deviceProp.totalGlobalMem;
  log_info("Mem: %uMB", gpu_mem / 1024 / 1024);
}

int main() {
  int c;
  int *dev_c;
  hipMalloc((void **)&dev_c, sizeof(int));
  add<<<1, 1>>>(2, 7, dev_c);
  hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
  log_info("2 + 7 = %d", c);
  hipFree(dev_c);
  init_gpu();
  return 0;
}
